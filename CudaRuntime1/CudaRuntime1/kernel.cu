#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <ctime>
#include <cstdio>
#include <random>
#define TTMATH_NOASM
#include "ttmath.h"
using namespace std;
typedef ttmath::Big<TTMATH_BITS(64), TTMATH_BITS(256)> lf;
typedef double ld;
typedef long long ll;
#define rnd(mt) ((mt=mt*1487267231ull+1499940803ull)%4999)
lf sqrt(lf x) {
	lf low = 0, high = 1; if (x > 1)high = x;
	for (int i = 0; i < 256; i++) {
		lf mid = (low + high) / 2;
		if (mid * mid <= x)
			low = mid;
		else
			high = mid;
	}
	return low;
}
#define AP 1887602657ull
#define P 1989944597ull
hipError_t addWithCuda(int *crr,int num);
__global__ void addKernel(int *crr,int num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	crr+= tid*10; crr[0] = 0;
	unsigned long long mt = AP+num*P+tid;
	for (ll iiii= 0;iiii<500000; iiii++) {
		#define r rnd(mt)
		ll Ax = 1000000-r, Ay= r-1000000, Az= r-r;
		ll Bx = r-r,By=1000000-r,Bz=r-1000000;
		ll Hx=Ay*Bz-By*Az,Hy=Az*Bx-Bz*Ax,Hz=Ax*By-Bx*Ay;
		#undef r
		
		ll Gx=Hx-Hy,Gy=Hy-Hz,Gz=Hz-Hx;
		ll Xa=1,Xb=-1,Xc=0; //<- little
		ll Ya=0,Yb=1,Yc=-1; //<- little
		ll Za=-1,Zb=0,Zc=1; //<- about 1e6
		if(abs(Gz)>1e3)
			continue;
		if(!Gx||!Gz)
			continue;
		ll Iz=970000;ll J=Iz*Gz;
		ll Ix=-J/Gx;J+=Ix*Gx;
		ll Iy=0;
		Iz-=J/Gz;J=Iz*Gz+Ix*Gx;
		if(J<0)J=-J;
		//if(J==0||J>100)
		//	continue;
		if(crr[0]&&crr[0]<=J)
			continue;
		crr[0] = J;
		crr[1] = Ax; crr[2] = Ay; crr[3] = Az;
		crr[4] = Bx; crr[5] = By; crr[6] = Bz;
		crr[7] = Ix-Iz; crr[8] = Iy-Ix; crr[9] = Iz-Iy;
	}
}

int main() 
{
	ld maxk = 0;
	ld mink = 1;
	int mtmp[10] = { 0 };
	int kkn = 0;
	static int crr[16384*10];
	for(int iiii=0;;iiii++){
		if (iiii % 30 == 0) {
			system("cls");
			cout << iiii << ':' << maxk << '/' << kkn << '=' << maxk / kkn << ';'<<mink << endl;
			for (int i = 0; i < 10; i++)
				cout << mtmp[i] << ' ';
			cout << endl;
			ld ex = iiii * maxk / kkn / 1.5e-19;
			cout << "remnant expectation: " << ex <<endl;
			cout << "property: " << pow(0.5, iiii / ex) << endl;
		}
		hipError_t cudaStatus = addWithCuda(crr,iiii);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
		cout<<"---";
		for(int iii=0;iii<16384;iii++)
			if(crr[iii*10]){
				ld k; {
					ll a=crr[iii * 10+1],b=crr[iii * 10+2],c=crr[iii * 10+3];
					ll d=crr[iii * 10+4],e=crr[iii * 10+5],f=crr[iii * 10+6];
					ll g=crr[iii * 10+7],h=crr[iii * 10+8],i=crr[iii * 10+9];
					lf AA = a * a + b * b + c * c; AA = sqrt(AA);
					lf BB = d * d + e * e + f * f; BB = sqrt(BB);
					lf CC = g * g + h * h + i * i; CC = sqrt(CC);
					lf ax = lf(a) / AA, ay = lf(b) / AA, az = lf(c) / AA;
					lf bx = lf(d) / BB, by = lf(e) / BB, bz = lf(f) / BB;
					lf cx = lf(g) / CC, cy = lf(h) / CC, cz = lf(i) / CC;
					ax -= cx; ay -= cy; az -= cz;
					bx -= cx; by -= cy; bz -= cz;
					lf dx = ay * bz - az * by;
					lf dy = az * bx - ax * bz;
					lf dz = ax * by - ay * bx;
					lf DD = sqrt(dx * dx + dy * dy + dz * dz);
					dx /= DD; dy /= DD; dz /= DD;
					k = abs((dx * cx + dy * cy + dz * cz).ToDouble());
				}
				cout << "!" << k; 
				if (k > 1e-30) {
					maxk = max(k, maxk);
					if (k < mink) {
						mink = k;
						memcpy(mtmp, crr + iii * 10, sizeof mtmp);
					}
					mink = min(k, mink);
					kkn++;
					if (k < 1.5e-19) {
						cout << endl;
						for (int j = 0; j < 10; j++)
							cout << crr[iii * 10 + j] << ' ';
						cout << endl;
						return 0;
					}
				}
			}
		cout << endl;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int num)
{
    int *dev_c = 0;
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_c, 16384*10*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    addKernel<<<128,128>>>(dev_c,num);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    cudaStatus = hipMemcpy(c, dev_c, 16384 * 10 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_c);
    return cudaStatus;
}
